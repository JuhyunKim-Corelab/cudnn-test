
// cudNNTest.cpp : Defines the entry point for the console application.
//
// Warning: Use at your own risk.

#include "stdafx.h"
#include "<your path here>\cudnn-6.5-win-R1\cudnn-6.5-win-R1\hipDNN.h"


int _tmain(int argc, _TCHAR* argv[])
{
hipdnnHandle_t hCudNN = NULL;
cudnnTensor4dDescriptor_t pInputDesc = NULL;
hipdnnFilterDescriptor_t pFilterDesc = NULL;
hipdnnConvolutionDescriptor_t pConvDesc = NULL;
cudnnTensor4dDescriptor_t pOutputDesc = NULL;
hipdnnStatus_t status;
hipError_t err;
int n_in = 64; // Number of images - originally 128
int c_in = 96; // Number of feature maps per image - originally 96
int h_in = 221; // Height of each feature map - originally 221
int w_in = 221; // Width of each feature map - originally 221
int k_pFilter_in = 256; // Number of output feature maps - originally 256
int c_pFilter_in = c_in; // Number of input feature maps - originally 96
int h_pFilter_in = 7; // Height of each pFilter - originally 7
int w_pFilter_in = 7; // Width of each pFilter - originally 7
int n_out = 0; // Number of output images.
int c_out = 0; // Number of output feature maps per image.
int h_out = 0; // Height of each output feature map.
int w_out = 0; // Width of each output feature map.

/* to change to double, chance HIPDNN_DATA_FLOAT to HIPDNN_DATA_DOUBLE and change each float to double below */

hipdnnDataType_t dataType = HIPDNN_DATA_FLOAT;
int nDataTypeSize = (((int)dataType)+1) * sizeof(float);
float* pImageInBatch = NULL;
float* pFilter = NULL;
float* pImageOutBatch = NULL;


try
{
//---------------------------------------
// Create CudNN
//---------------------------------------

status = hipdnnCreate(&hCudNN);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


//---------------------------------------
// Create Descriptors
//---------------------------------------

status = cudnnCreateTensor4dDescriptor(&pInputDesc);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;

status = cudnnCreateTensor4dDescriptor(&pOutputDesc);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;

status = hipdnnCreateFilterDescriptor(&pFilterDesc);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;

status = hipdnnCreateConvolutionDescriptor(&pConvDesc);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


//---------------------------------------
// Allocate memory for pFilter and ImageBatch
//---------------------------------------

err = hipMalloc((void**)&pImageInBatch, n_in*c_in*h_in*w_in * nDataTypeSize);
if (err != hipSuccess)
throw err;

err = hipMalloc((void**)&pFilter , k_pFilter_in*c_pFilter_in*h_pFilter_in*w_pFilter_in * nDataTypeSize);
if (err != hipSuccess)
throw err;


//---------------------------------------
// Fill the input image and pFilter data
//---------------------------------------

//TODO: Still figuring this out


//---------------------------------------
// Set decriptors
//---------------------------------------

status = hipdnnSetTensor4dDescriptor(pInputDesc, HIPDNN_TENSOR_NCHW, dataType, n_in, c_in, h_in, w_in);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;

status = cudnnSetFilterDescriptor(pFilterDesc, dataType, k_pFilter_in, c_pFilter_in, h_pFilter_in, w_pFilter_in);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;

//HIPDNN_CROSS_CORRELATION
status = cudnnSetConvolutionDescriptor(pConvDesc, pInputDesc, pFilterDesc, 0, 0, 2, 2, 1, 1, HIPDNN_CONVOLUTION);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


//---------------------------------------
// Query output layout
//---------------------------------------

status = cudnnGetOutputTensor4dDim(pConvDesc, CUDNN_CONVOLUTION_FWD, &n_out, &c_out, &h_out, &w_out);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


//---------------------------------------
// Set and allocate output tensor descriptor
//---------------------------------------

status = hipdnnSetTensor4dDescriptor(pOutputDesc, HIPDNN_TENSOR_NCHW, dataType, n_out, c_out, h_out, w_out);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


err = hipMalloc((void**)&pImageOutBatch, n_out*c_out*h_out*w_out * nDataTypeSize);
if (err != hipSuccess)
throw err;


//---------------------------------------
// Launch convolution on GPU
//---------------------------------------

status = hipdnnConvolutionForward(hCudNN, pInputDesc, pImageInBatch, pFilterDesc, pFilter, pConvDesc, pOutputDesc, pImageOutBatch, CUDNN_RESULT_NO_ACCUMULATE);
if (status != HIPDNN_STATUS_SUCCESS)
throw status;


//---------------------------------------
// Extract output data
//---------------------------------------

//TBD
}
catch (...)
{
}

//---------------------------------------
// Clean-up
//---------------------------------------

if (pImageInBatch != NULL)
hipFree(pImageInBatch);

if (pImageOutBatch != NULL)
hipFree((void*)pImageOutBatch);

if (pFilter != NULL)
hipFree((void*)pFilter);

if (pInputDesc != NULL)
cudnnDestroyTensor4dDescriptor(pInputDesc);

if (pOutputDesc != NULL)
cudnnDestroyTensor4dDescriptor(pOutputDesc);

if (pFilterDesc != NULL)
hipdnnDestroyFilterDescriptor(pFilterDesc);

if (pConvDesc != NULL)
hipdnnDestroyConvolutionDescriptor(pConvDesc);

if (hCudNN != NULL)
hipdnnDestroy(hCudNN);

return 0;
} 